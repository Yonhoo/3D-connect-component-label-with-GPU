#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cuda_occupancy.h>
#define RAD 1		//local block radius
#define TX 32		//number of threads per block along x-axis 
#define TY 32		//number of threads per block along y-axis
#define TZ 1		//number of threads per block along z-axis

using namespace std;




__device__ int diff(int d1, int d2)
{
	return abs(((d1 >> 16) & 0xff) - ((d2 >> 16) & 0xff)) + abs(((d1 >> 8) & 0xff) - ((d2 >> 8) & 0xff)) + abs((d1 & 0xff) - (d2 & 0xff));
}

//this is Prevent cross-border
__device__ int idxClip(int idx, int idxMax)
{
	//Prevent cross-border
	return idx > (idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx);
}

__device__ int flatten(int col, int row, int slice, int w, int h, int z)
{
	return  idxClip(col, w) + idxClip(row, h)*w + idxClip(slice, z)*w*h;
}


//not use share memory
__global__ void init_CCL_26(int *L,
	int * R,
	int w,
	int h,
	int z)
{
	const int col = blockIdx.x*blockDim.x + threadIdx.x;
	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int sli = blockIdx.z*blockDim.z + threadIdx.z;
	if ((col >= w) || (row >= h) || (sli >= z))	return;
	const int id = flatten(col, row, sli, w, h, z);
	L[id] = R[id] = id;
}

inline
__device__ void Host2share_26(const uint3 threadx,
	const dim3 blockdim,
	int * origin_DATA,
	int *s_in,
	const int *s_l,
	const int *s_i,
	const int *g_l,
	const int *g_i)
{
	// Resolving overlapping parts
	if (threadIdx.x < RAD&&threadIdx.y < RAD)
	{
		s_in[flatten(s_i[0] - RAD, s_i[1] - RAD, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1] - RAD, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] - RAD, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] - RAD, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] + blockDim.y, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1] + blockDim.y, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] + blockDim.y, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] + blockDim.y, g_i[2], g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.x < RAD&&threadIdx.z < RAD)
	{
		s_in[flatten(s_i[0] - RAD, s_i[1], s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1], g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1], s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1], g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1], s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1], g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1], s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1], g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.y < RAD&&threadIdx.z < RAD)
	{
		s_in[flatten(s_i[0], s_i[1] - RAD, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] - RAD, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] + blockDim.y, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] + blockDim.y, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] - RAD, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] - RAD, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] + blockDim.y, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] + blockDim.y, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.x < RAD) {
		s_in[flatten(s_i[0] - RAD, s_i[1], s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1], g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1], s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1], g_i[2], g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.y < RAD) {
		s_in[flatten(s_i[0], s_i[1] - RAD, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] - RAD, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] + blockDim.y, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] + blockDim.y, g_i[2], g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.z < RAD) {
		s_in[flatten(s_i[0], s_i[1], s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1], g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1], s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1], g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.x < RAD&&threadIdx.y < RAD&&threadIdx.z < RAD) {
		s_in[flatten(s_i[0] - RAD, s_i[1] - RAD, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] 
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1] - RAD, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] - RAD, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1] - RAD, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] + blockDim.y, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1]+blockDim.y, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] + blockDim.y, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1] + blockDim.y, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] - RAD, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] - RAD, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] + blockDim.y, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] + blockDim.y, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] + blockDim.y, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] + blockDim.y, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] - RAD, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] - RAD, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
}

inline
__device__ void Host2share_26_C(const uint3 threadx,
	const dim3 blockdim,
	uchar * origin_DATA,
	uchar *s_in,
	const int *s_l,
	const int *s_i,
	const int *g_l,
	const int *g_i)
{
	// Resolving overlapping parts
	if (threadIdx.x < RAD&&threadIdx.y < RAD)
	{
		s_in[flatten(s_i[0] - RAD, s_i[1] - RAD, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1] - RAD, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] - RAD, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] - RAD, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] + blockDim.y, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1] + blockDim.y, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] + blockDim.y, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] + blockDim.y, g_i[2], g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.x < RAD&&threadIdx.z < RAD)
	{
		s_in[flatten(s_i[0] - RAD, s_i[1], s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1], g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1], s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1], g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1], s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1], g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1], s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1], g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.y < RAD&&threadIdx.z < RAD)
	{
		s_in[flatten(s_i[0], s_i[1] - RAD, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] - RAD, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] + blockDim.y, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] + blockDim.y, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] - RAD, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] - RAD, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] + blockDim.y, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] + blockDim.y, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.x < RAD) {
		s_in[flatten(s_i[0] - RAD, s_i[1], s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] - RAD, g_i[1], g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1], s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1], g_i[2], g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.y < RAD) {
		s_in[flatten(s_i[0], s_i[1] - RAD, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] - RAD, g_i[2], g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1] + blockDim.y, s_i[2], s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1] + blockDim.y, g_i[2], g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.z < RAD) {
		s_in[flatten(s_i[0], s_i[1], s_i[2] - RAD, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1], g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0], s_i[1], s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])] =
			origin_DATA[flatten(g_i[0], g_i[1], g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
	if (threadIdx.x < RAD&&threadIdx.y < RAD&&threadIdx.z < RAD) {
		s_in[flatten(s_i[0] - RAD, s_i[1] - RAD, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1] - RAD, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] - RAD, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1] - RAD, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] + blockDim.y, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1] + blockDim.y, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] - RAD, s_i[1] + blockDim.y, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] - RAD, g_i[1] + blockDim.y, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] - RAD, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] - RAD, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] + blockDim.y, s_i[2] - RAD, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] + blockDim.y, g_i[2] - RAD, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] + blockDim.y, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] + blockDim.y, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
		s_in[flatten(s_i[0] + blockDim.x, s_i[1] - RAD, s_i[2] + blockDim.z, s_l[0], s_l[1], s_l[2])]
			= origin_DATA[flatten(g_i[0] + blockDim.x, g_i[1] - RAD, g_i[2] + blockDim.z, g_l[0], g_l[1], g_l[2])];
	}
}

inline
__device__ void min_26_nbd(int data,
	uchar* s_in,
	int* L_label,
	const int s_w,
	int *label,
	const int s_index,
	const int  slice_area,
	const int elpise)
{
	/*--------------current slice-----------------------*/
	//up
	if (diff(data, s_in[s_index - s_w]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - s_w]);
	//down
	if (diff(data, s_in[s_index + s_w]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + s_w]);
	//left
	if (diff(data, s_in[s_index - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - 1]);
	//right
	if (diff(data, s_in[s_index + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + 1]);
	//up left
	if (diff(data, s_in[s_index - s_w - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - s_w - 1]);
	//up right
	if (diff(data, s_in[s_index - s_w + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - s_w + 1]);
	//down left
	if (diff(data, s_in[s_index + s_w - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + s_w - 1]);
	//down right
	if (diff(data, s_in[s_index + s_w + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + s_w + 1]);
	/*--------------up slice-----------------------*/
	//cur index
	if (diff(data, s_in[s_index - slice_area]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area]);
	//up
	if (diff(data, s_in[s_index - slice_area - s_w]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area - s_w]);
	//down
	if (diff(data, s_in[s_index - slice_area + s_w]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area + s_w]);
	//left
	if (diff(data, s_in[s_index - slice_area - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area - 1]);
	//right
	if (diff(data, s_in[s_index - slice_area + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area + 1]);
	//up left
	if (diff(data, s_in[s_index - slice_area - s_w - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area - s_w - 1]);
	//up right
	if (diff(data, s_in[s_index - slice_area - s_w + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area - s_w + 1]);
	//down left
	if (diff(data, s_in[s_index - slice_area + s_w - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area + s_w - 1]);
	//down right
	if (diff(data, s_in[s_index - slice_area + s_w + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index - slice_area + s_w + 1]);
	/*--------------down slice-----------------------*/
	//cur index
	if (diff(data, s_in[s_index + slice_area]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area]);
	//up
	if (diff(data, s_in[s_index + slice_area - s_w]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area - s_w]);
	//down
	if (diff(data, s_in[s_index + slice_area + s_w]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area + s_w]);
	//left
	if (diff(data, s_in[s_index + slice_area - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area - 1]);
	//right
	if (diff(data, s_in[s_index + slice_area + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area + 1]);
	//up left
	if (diff(data, s_in[s_index + slice_area - s_w - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area - s_w - 1]);
	//up right
	if (diff(data, s_in[s_index + slice_area - s_w + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area - s_w + 1]);
	//down left
	if (diff(data, s_in[s_index + slice_area + s_w - 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area + s_w - 1]);
	//down right
	if (diff(data, s_in[s_index + slice_area + s_w + 1]) <= elpise)
		*label = min((int)*label, (int)L_label[s_index + slice_area + s_w + 1]);
	/*----------------above------------------------*/
}

inline
__device__ void min_26_nbd_control_bound(
	const int g_l[3],
	const int g_in[3],
	int data,
	uchar* s_in,
	int* L_label,
	const int s_w,
	int *label,
	const int s_index,
	const int  slice_area,
	const int elpise)
{
	/*--------------current slice-----------------------*/
	if (g_in[1] > 0) {
		//up
		if (diff(data, s_in[s_index - s_w]) <= elpise)
			*label = min((int)*label, (int)L_label[s_index - s_w]);
		if (g_in[0] > 0)
		{
			//up left
			if (diff(data, s_in[s_index - s_w - 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index - s_w - 1]);
		}
		if (g_in[0] < g_l[0] - 1)
		{
			//up right
			if (diff(data, s_in[s_index - s_w + 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index - s_w + 1]);
		}

	}
	if (g_in[1] < g_l[1] - 1) {
		//down
		if (diff(data, s_in[s_index + s_w]) <= elpise)
			*label = min((int)*label, (int)L_label[s_index + s_w]);
		if (g_in[0] > 0)
		{
			//down left
			if (diff(data, s_in[s_index + s_w - 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index + s_w - 1]);
		}
		if (g_in[0] < g_l[0] - 1)
		{
			//down right
			if (diff(data, s_in[s_index + s_w + 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index + s_w + 1]);
		}
	}
	if (g_in[0] > 0) {
		//left
		if (diff(data, s_in[s_index - 1]) <= elpise)
			*label = min((int)*label, (int)L_label[s_index - 1]);
	}
	if (g_in[0] < g_l[0] - 1) {
		//right
		if (diff(data, s_in[s_index + 1]) <= elpise)
			*label = min((int)*label, (int)L_label[s_index + 1]);
	}
	
	
	/*--------------up slice-----------------------*/
	if (g_in[2] > 0)
	{
		//cur index
		if (diff(data, s_in[s_index - slice_area]) <= elpise)
			*label = min((int)*label, (int)L_label[s_index - slice_area]);
		if (g_in[1] > 0) 
		{
			//up
			if (diff(data, s_in[s_index - slice_area - s_w]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index - slice_area - s_w]);
			if (g_in[0] > 0)
			{
				//up left
				if (diff(data, s_in[s_index - slice_area - s_w - 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index - slice_area - s_w - 1]);
			}
			if (g_in[0] < g_l[0] - 1)
			{
				//up right
				if (diff(data, s_in[s_index - slice_area - s_w + 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index - slice_area - s_w + 1]);
			}
		}
		if (g_in[1] < g_l[1] - 1) {
			//down
			if (diff(data, s_in[s_index - slice_area + s_w]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index - slice_area + s_w]);
			if (g_in[0] > 0)
			{
				//down left
				if (diff(data, s_in[s_index - slice_area + s_w - 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index - slice_area + s_w - 1]);
			}
			if (g_in[0] < g_l[0] - 1)
			{
				//down right
				if (diff(data, s_in[s_index - slice_area + s_w + 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index - slice_area + s_w + 1]);
			}

		}
		if (g_in[0] > 0) {
			//left
			if (diff(data, s_in[s_index - slice_area - 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index - slice_area - 1]);
		}
		if (g_in[0] < g_l[0] - 1) {
			//right
			if (diff(data, s_in[s_index - slice_area + 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index - slice_area + 1]);
		}
		
	}
	
	/*--------------down slice-----------------------*/
	if (g_in[2] < g_l[2] - 1)
	{
		//cur index
		if (diff(data, s_in[s_index + slice_area]) <= elpise)
			*label = min((int)*label, (int)L_label[s_index + slice_area]);
		if (g_in[1] > 0)
		{
			//up
			if (diff(data, s_in[s_index + slice_area - s_w]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index + slice_area - s_w]);
			if (g_in[0] > 0)
			{
				//up left
				if (diff(data, s_in[s_index + slice_area - s_w - 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index + slice_area - s_w - 1]);
			}
			if (g_in[0] < g_l[0] - 1)
			{
				//up right
				if (diff(data, s_in[s_index + slice_area - s_w + 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index + slice_area - s_w + 1]);
			}
		}
		if (g_in[1] < g_l[1] - 1)
		{
			//down
			if (diff(data, s_in[s_index + slice_area + s_w]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index + slice_area + s_w]);
			if (g_in[0] > 0)
			{
				//down left
				if (diff(data, s_in[s_index + slice_area + s_w - 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index + slice_area + s_w - 1]);
			}
			if (g_in[0] < g_l[0] - 1)
			{
				//down right
				if (diff(data, s_in[s_index + slice_area + s_w + 1]) <= elpise)
					*label = min((int)*label, (int)L_label[s_index + slice_area + s_w + 1]);
			}
		}
		if (g_in[0] > 0)
		{
			//left
			if (diff(data, s_in[s_index + slice_area - 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index + slice_area - 1]);
		}
		if (g_in[0] < g_l[0] - 1)
		{
			//right
			if (diff(data, s_in[s_index + slice_area + 1]) <= elpise)
				*label = min((int)*label, (int)L_label[s_index + slice_area + 1]);
		}
			
	}
	/*----------------above------------------------*/
}


//3d
__global__
void scanning26(unsigned char* origin_DATA,
	int* L_label,
	int* R_label,
	int w,
	int h,
	int z,
	bool *check)
{
	const int elpise = 1E-4;
	//1000
	__shared__ uchar s_in[(TX + 2)*(TY + 2)*(TZ + 2)];
	__shared__ int s_lb[(TX + 2)*(TY + 2)*(TZ + 2)];

	//global index
	
	const int col = blockIdx.x*blockDim.x + threadIdx.x;
	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int sli = blockIdx.z*blockDim.z + threadIdx.z;
	if ((col >= w) || (row >= h) || (sli >= z))	return;
	const int id = flatten(col, row, sli, w, h, z);
	int label = w*h*z;

	const int g_local[3] = { w ,h,z };
	const int g_index[3] = { col ,row,sli };


	//local width and height
	const int s_w = blockDim.x + 2 * RAD;
	const int s_h = blockDim.y + 2 * RAD;
	const int s_z = blockDim.z + 2 * RAD;
	const int slice_area = s_w*s_h;



	//local index
	const int s_col = threadIdx.x + RAD;
	const int s_row = threadIdx.y + RAD;
	const int s_sli = threadIdx.z + RAD;
	const int s_index = flatten(s_col, s_row, s_sli, s_w, s_h, s_z);

	const int s_local[3] = { s_w ,s_h,s_z };
	const int s_ind[3] = { s_col ,s_row,s_sli };
	//global data to share block data
	s_in[s_index] = origin_DATA[id];
	s_lb[s_index] = L_label[id];

	//host data to share
	Host2share_26_C(threadIdx, blockDim, origin_DATA, s_in, s_local, s_ind, g_local, g_index);
	Host2share_26(threadIdx, blockDim, L_label, s_lb, s_local, s_ind, g_local, g_index);

	__syncthreads();
	
	
	//current data
	int data = s_in[s_index];
	__syncthreads();
	//find neighbor min label
	min_26_nbd(data, s_in, s_lb, s_w, &label, s_index, slice_area, elpise);
	__syncthreads();


	if (label < s_lb[s_index]) {
		//atomicMin(&R[L[id]], label);
		R_label[s_lb[s_index]] = label;				//修改的是
		*check = true;
	}
}


__global__ void analysis26(int* L_label,
	int* R_label,
	int w,
	int h,
	int z)

{
	//there is not using share memory
	//global index
	const int col = blockIdx.x*blockDim.x + threadIdx.x;
	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int sli = blockIdx.z*blockDim.z + threadIdx.z;
	if ((col >= w) || (row >= h) || (sli >= z))	return;
	const int id = flatten(col, row, sli, w, h, z);

	int label = L_label[id];
	int ref;
	if (label == id) {
		//找到它的局部最小值因为当右边R矩阵记录了领域最小，那么它就会记录上一个最小的最小
		//因此我一直迭代，就找到了局部最小值
		do{ 
			label = R_label[ref = label]; 
		}while (ref ^ label);
		R_label[id] = label;
	}
}

inline
int divUp(int a, int b)
{
	return (a + b - 1) / b;
}

__global__ void labeling26(int* L_label,
	int* R_label,
	int w,
	int h,
	int z)
{
	//global index
	const int col = blockIdx.x*blockDim.x + threadIdx.x;
	const int row = blockIdx.y*blockDim.y + threadIdx.y;
	const int sli = blockIdx.z*blockDim.z + threadIdx.z;
	if ((col >= w) || (row >= h) || (sli >= z))	return;
	const int id = flatten(col, row, sli, w, h, z);

	//此时L[id]的标签还是L[id]，就还是当前最小
	L_label[id] = R_label[R_label[L_label[id]]];
}



int * CCL::cuda_ccl(unsigned char * image,
	int degree_of_connectivity,
	int threshold,
	const int WIDTH,
	const int HEIGHT,
	const int SLICE)
{
	const int Elem = WIDTH*HEIGHT*SLICE;
	hipError_t Error;
	int * result = (int *)malloc(Elem * sizeof(int));
	uchar *origin_data;
	int *L_label, *R_label;
	Error = hipMalloc(&origin_data, sizeof(int) * Elem);
	if (Error != hipSuccess)
		cout << "origin_data hipMalloc error" << endl;
	Error = hipMalloc(&L_label, sizeof(int) * Elem);
	if (Error != hipSuccess)
		cout << "L_label hipMalloc error" << endl;
	Error = hipMalloc(&R_label, sizeof(int) * Elem);
	if (Error != hipSuccess)
		cout << "R_label hipMalloc error" << endl;

	Error = hipMemcpy(origin_data, image, sizeof(unsigned char) * Elem, hipMemcpyHostToDevice);
	if (Error != hipSuccess)
		cout << "hipMemcpy error" << endl;

	bool* md;
	Error = hipMalloc((void**)&md, sizeof(bool));
	if (Error != hipSuccess)
		cout << "hipMemcpy error" << endl;
	int blocks;
	int grids;
	dim3 block(TX, TY,TZ);	
	dim3 grid(divUp(WIDTH, TX), divUp(HEIGHT, TY), divUp(SLICE, TZ));
	
	init_CCL_26 << <grid, block >> > (L_label, R_label, WIDTH, HEIGHT, SLICE);
	hipDeviceSynchronize();
	auto err = hipGetLastError();
	if (err != hipSuccess)
		cout << "error" << endl;
	
	const size_t smSz = (TX + 2 * RAD)*(TY + 2 * RAD) *(TZ + 2 * RAD) * sizeof(int);
	for (;;) {
		bool m = false;
		Error = hipMemcpy(md, &m, sizeof(bool), hipMemcpyHostToDevice);
		if (Error != hipSuccess) {
			cout << "hipMemcpy error" << endl;
		}
		scanning26 << <grid, block >> > (origin_data, L_label, R_label, WIDTH, HEIGHT, SLICE, md);
		err = hipGetLastError();
		if (err != hipSuccess)
			cout << "error" << endl;
		hipMemcpy(&m, md, sizeof(bool), hipMemcpyDeviceToHost);
		if (Error != hipSuccess) {
			cout << "hipMemcpy error" << endl;
		}
		if (m) {
			analysis26 << <grid, block >> >(L_label, R_label, WIDTH, HEIGHT, SLICE);
			err = hipGetLastError();
			if (err != hipSuccess)
				cout << "error" << endl;
			//hipDeviceSynchronize();
			labeling26 << <grid, block >> >(L_label, R_label, WIDTH, HEIGHT, SLICE);
			err = hipGetLastError();
			if (err != hipSuccess)
				cout << "error" << endl;
		}
		else break;
	}

	hipMemcpy(result, L_label, Elem * sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(L_label);
	hipFree(R_label);
	hipFree(origin_data);
	

	return result;
}
